#include <stdio.h>
#include "hip/hip_runtime.h"



int threshold = 30;

__global__ void kernel(int *img, int *result, int rows, int cols, int threshold)
{
    //int i = (blockIdx.x*blockDim.x)+threadIdx.x;

    int i = blockIdx.x+1;
    int j = threadIdx.x+1;
    int k,l;


    // int i,j,k,l;

    int t = threshold;
    int px,gx,gy,ox,oy;

    px = 0;
    gx = 0; gy = 0;
    ox = 0; oy = 0;

    for (k=i-1; k<=i+1; k++) {
        for (l=j-1; l<=j+1; l++) {
            ox = (l-j) * (k==i ? 2:1);
            oy = (i-k) * (l==j ? 2:1);

            gx += img[k*cols+l]*ox;
            gy += img[k*cols+l]*oy;
        }
    }

    px = gx<0 ? -gx:gx + gy<0 ? -gy:gy;
    px = px>t ? 255 : 0;
    result[(i-1)*(cols-2)+(j-1)] = px;


}




void cuda(int *img, int *result, int rows, int cols) {

    int i,j,k,l;

    int t = threshold;
    int px,gx,gy,ox,oy;

    for(i = 1; i < rows-1; i++)
    {
        for(j = 1; j < cols-1; j++)
        {
            px = 0;
            gx = 0; gy = 0;
            ox = 0; oy = 0;

            for (k=i-1; k<=i+1; k++) {
                for (l=j-1; l<=j+1; l++) {
                    ox = (l-j) * (k==i ? 2:1);
                    oy = (i-k) * (l==j ? 2:1);

                    gx += img[k*cols+l]*ox;
                    gy += img[k*cols+l]*oy;
                }
            }

            px = gx<0?-gx:gx + gy<0?-gy:gy;
            px = px>t ? 255 : 0;
            result[(i-1)*(cols-2)+(j-1)] = px;
        }
    }


}

int main() {

    FILE *pFile = fopen ("input.txt","r");

    int imgRow,imgCol;
    int i,j;

    fscanf(pFile, "%d %d", &imgRow, &imgCol);
    int *img = (int*) malloc(imgRow*imgCol*sizeof(int));
    for(i = 0; i < imgRow; i++)
    {
        for(j = 0; j < imgCol; j++)
        {
            fscanf(pFile, "%d", &img[i*imgCol+j]);
        }
        printf("read file:%d/%d\n",i+1,imgRow);
    }
    fclose(pFile);



    int *result = (int*) malloc(imgRow*imgCol*sizeof(int));

    int k,l;
    int *temp;


    int maxDataPerRow = 256;
    int maxDataPerCol = 256;


    // Splite work by rows
    int secRow = ( (imgRow+1) / (maxDataPerRow-2) ) ;
    int *secSizeRow  = (int*) malloc(secRow*sizeof(int));
    int *secDisplRow = (int*) malloc((secRow+1)*sizeof(int));
    l = 0; i = 0;
    while( l<imgRow ) {
        secDisplRow[i] = l;
        l += maxDataPerRow;
        if(l>=imgRow)
            break;
        secSizeRow[i] = l - secDisplRow[i];
        l -= 2; i++;
    }

    l = imgRow;
    secSizeRow[i] = l - secDisplRow[i];
    secDisplRow[i+1] = secDisplRow[i] + secSizeRow[i]-4;



    // Splite work by cols
    int secCol = ( (imgCol+1) / (maxDataPerCol-2) ) ;
    int *secSizeCol  = (int*) malloc(secCol*sizeof(int));
    int *secDisplCol = (int*) malloc((secCol+1)*sizeof(int));
    l = 0; i = 0;
    while( l<imgCol ) {
        secDisplCol[i] = l;
        l += maxDataPerCol;
        if(l>=imgCol)
            break;
        secSizeCol[i] = l - secDisplCol[i];
        l -= 2; i++;
    }

    l = imgCol;
    secSizeCol[i] = l - secDisplCol[i];
    secDisplCol[i+1] = secDisplCol[i] + secSizeCol[i]-4;


    for (i=0; i<=secRow; i++) {
        for (j=0; j<=secCol; j++) {

            int *array = (int*) malloc((secSizeRow[i]*secSizeCol[j])*sizeof(int));
            temp = &array[0];
            for(k=secDisplRow[i]; k<secDisplRow[i]+secSizeRow[i]; k++) {
                for(l=secDisplCol[j]; l<secDisplCol[j]+secSizeCol[j]; l++) {
                    *temp = img[k*imgCol+l];
                    temp++;
                }
            }


            int *array_dev;
            hipMalloc((void**) &array_dev, (secSizeRow[i]*secSizeCol[j])*sizeof(int));

            hipMemcpy(array_dev,array,(secSizeRow[i]*secSizeCol[j])*sizeof(int),hipMemcpyHostToDevice);


            // Result

            int *cudaResult = (int*) malloc( (secSizeRow[i]-2) * (secSizeCol[j]-2)*sizeof(int) );

            int *cudaResult_dev;
            hipMalloc((void**) &cudaResult_dev, (secSizeRow[i]-2) * (secSizeCol[j]-2)*sizeof(int));


            int threadNum = secSizeRow[i]-2;
            int blockNum  = secSizeCol[j]-2;


            kernel <<<blockNum,threadNum>>>(array_dev, cudaResult_dev, secSizeRow[i], secSizeCol[j], threshold);


            hipMemcpy(cudaResult,cudaResult_dev,(secSizeRow[i]-2) * (secSizeCol[j]-2)*sizeof(int),hipMemcpyDeviceToHost);



            temp = &cudaResult[0];
            for(k=secDisplRow[i]+1; k<secDisplRow[i]+secSizeRow[i]-1; k++) {
                for(l=secDisplCol[j]+1; l<secDisplCol[j]+secSizeCol[j]-1; l++) {
                    result[k*imgCol+l] = *temp;
                    temp++;
                }
            }


            free(cudaResult);
            free(array);

            printf("piece:%d,%d\n",i,j);

        }
    }

    free(img);

    pFile = fopen ("output.txt","w");
    fprintf(pFile, "%d %d\n", imgRow, imgCol);

    for(i = 0; i < imgRow; i++)
    {
        for(j = 0; j < imgCol; j++)
        {
            if(i==0 || i==imgRow-1 || j==0 || j==imgRow-1)
                result[i*imgCol+j] = 0;

            fprintf(pFile, "%d ", result[i*imgCol+j]);
            // if(result[i*imgCol+j]>0)
            //     printf("!");

        }
        fprintf(pFile, "\n");
        printf("write file:%d/%d\n",i+1,imgRow);
    }

    fclose(pFile);

    free(result);



    return 0;

}



